
extern "C" {
#include "physics.h"
}

#include "solve.cu"

#include <stdio.h>

#include <hiprand.h>


#define RESERVE 2000


static size_t reserved = 0;

static float* d_balls = NULL;


void reserve(size_t size)
{
    reserved = size;

    if (d_balls != NULL)
    {
        hipFree(d_balls);
    }

    if (hipMalloc(&d_balls, size * SIZE * sizeof(float)) != hipSuccess)
    {
        fprintf(stderr, "(CUDA) ERROR: allocation failed\n");
    }
}


extern "C"
int physics_init()
{
    const int device = 0;

    {
        hipError_t err = hipSetDevice(device);
        if (err != hipSuccess)
        {
            fprintf(stderr, "(CUDA) ERROR: cannot set CUDA device: %s\n",
                            hipGetErrorString(err));
            return 1;
        }
    }

    {
        void* ptr = NULL;
        hipError_t err = hipMalloc(&ptr, 1024);
        if (err != hipSuccess)
        {
            fprintf(stderr, "(CUDA) ERROR: %s\n", hipGetErrorString(err));
            return 1;
        }
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("(CUDA) Using device %d: '%s'\n", device, deviceProp.name);

    reserve(RESERVE);

    return 0;
}


extern "C"
void physics_quit()
{
    hipFree(d_balls);
}


extern "C"
void solve_circles(float* balls, int count,
                   float gravity,
                   float minx, float maxx,
                   float miny, float maxy)
{
    while (count > reserved)
    {
        reserve(2 * reserved);
    }

    hipMemcpy(d_balls, balls, count * SIZE * sizeof(float),
               hipMemcpyHostToDevice);

    solve_gpu(d_balls, count, gravity, minx, maxx, miny, maxy);

    hipMemcpy(balls, d_balls, count * SIZE * sizeof(float),
               hipMemcpyDeviceToHost);
}
